
#include <hip/hip_runtime.h>
#include <stdio.h>

__constant__ int const_array[4];

__global__ void kernel(int* array) {
  int tid = threadIdx.x;
  //array[tid] += const_array[tid];
  printf("%d, %d, %d\n ",tid,const_array[tid],array[tid]);
}

int main() {
  int array[4] = {1, 2, 3, 4};
  int size = 4 * sizeof(int);
  int* device_array;

  hipMalloc(&device_array, size);
  hipMemcpy(device_array, array, size, hipMemcpyHostToDevice);
  //cudaMemcpy(const_array, array, size, cudaMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(const_array), array, size);

  kernel<<<1, 4>>>(device_array);

  hipFree(device_array);
  return 0;
}

