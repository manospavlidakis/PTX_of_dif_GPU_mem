#include "hip/hip_runtime.h"
#define N 1024

// texture object is a kernel argument
__global__ void kernel(hipTextureObject_t tex, float *A) {
  int i = blockIdx.x *blockDim.x + threadIdx.x;
  float x = tex1Dfetch<float>(tex, i);
  A[i] = x; 
  // do some work using x ...
}

void call_kernel(hipTextureObject_t tex) {
  dim3 block(128,1,1);
  dim3 grid(N/block.x,1,1);
  //float *A;
  //hipMalloc(&A, 4*sizeof(float));

  kernel <<<grid, block>>>(tex);
}

int main() {
  // declare and allocate memory
  float *buffer;
  hipMalloc(&buffer, N*sizeof(float));

  // create texture object
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = buffer;
  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  resDesc.res.linear.desc.x = 32; // bits per channel
  resDesc.res.linear.sizeInBytes = N*sizeof(float);

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;

  // create texture object: we only have to do this once!
  hipTextureObject_t tex=0;
  hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

  call_kernel(tex); // pass texture as argument

  // destroy texture object
  hipDestroyTextureObject(tex);

  hipFree(buffer);
}
