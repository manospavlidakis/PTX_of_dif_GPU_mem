
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

typedef uint8_t mt;  // use an integer type

__global__ void kernel(hipTextureObject_t tex, int *outputData)
{
    int x = threadIdx.x;
    //int y = 98; //NO error!!!!
    int y = threadIdx.y;
    mt val = tex2D<mt>(tex, x, y);
    outputData[0] = val;
    

    //mt val = tex2D<mt>(tex, x, y);
//    printf("x=%d, y=%d, val=%d \n",x,y, val);
}

int main(int argc, char **argv)
{
    int *dData = NULL;
    hipMalloc((void **) &dData, 5);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("texturePitchAlignment: %lu\n", prop.texturePitchAlignment);
    hipTextureObject_t tex;
    const int num_rows = 4;
    const int num_cols = prop.texturePitchAlignment*1; // should be able to use a different multiplier here
    const int ts = num_cols*num_rows;
    const int ds = ts*sizeof(mt);
    mt dataIn[ds];
    for (int i = 0; i < ts; i++) dataIn[i] = i;
    mt* dataDev = 0;
    size_t pitch;
    hipMallocPitch((void**)&dataDev, &pitch,  num_cols*sizeof(mt), num_rows);
    hipMemcpy2D(dataDev, pitch, dataIn, num_cols*sizeof(mt), num_cols*sizeof(mt), num_rows, hipMemcpyHostToDevice);
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = dataDev;
    resDesc.res.pitch2D.width = num_cols;
    resDesc.res.pitch2D.height = num_rows;
    resDesc.res.pitch2D.desc = hipCreateChannelDesc<mt>();
    resDesc.res.pitch2D.pitchInBytes = pitch;
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
    dim3 threads(4, 4);
    kernel<<<1, threads>>>(tex, dData);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
          printf("Error launching kernel: %s\n", hipGetErrorString(err));
          return 1;
    }
    printf("\n");
    return 0;
}
